#include "hip/hip_runtime.h"
// -*- mode:c++;tab-width:2;indent-tabs-mode:t;show-trailing-whitespace:t;rm-trailing-spaces:t -*-
// vi: set ts=2 noet:
//
// (c) Copyright Rosetta Commons Member Institutions.
// (c) This file is part of the Rosetta software suite and is made available under license.
// (c) The Rosetta software is developed by the contributing members of the Rosetta Commons.
// (c) For more information, see http://www.rosettacommons.org. Questions about this can be
// (c) addressed to University of Washington CoMotion, email: license@uw.edu.

/// @file   core/scoring/fiber_diffractiobn/FiberDiffractionKernel.cu
/// @brief  FiberDiffraction GPU support
/// @author Wojtek Potrzebowski and Ingemar Andre

#include <utility/vector0.hh>
#include <utility/vector1.hh>

#include <numeric/xyzVector.hh>
#include <basic/Tracer.hh>

#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <core/types.hh>
#include <sys/time.h>

#include "cutil_math.h"

namespace core {
namespace scoring {
namespace fiber_diffraction {

static basic::Tracer TR("core.scoring.fiber_diffraction.FiberDiffractionKernelGpu");

__global__
void calculate_bessels_kernel(
	int l, 
	int n, 
	int abs_n, 
	int const natoms,
	int const legal_R_values,
	float * d_layer_lines_R,
	Real * d_r, float * d_bessel)
{
	unsigned int atom1 = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int R = blockDim.y * blockIdx.y + threadIdx.y;
	if (atom1 < natoms && R < legal_R_values) {
		Real Rinv(d_layer_lines_R[R]);
		Real x_factor( 2*M_PI*Rinv );
		Real X1 (x_factor*d_r[atom1]);
		if ( abs_n <= X1 +2 ) {
			if (n==0) d_bessel[natoms*R+atom1]=j0f(X1);
			if (n==1) d_bessel[natoms*R+atom1]=j1f(X1);
			if (n==-1) d_bessel[natoms*R+atom1]=-j1f(X1);
			if (n>1) d_bessel[natoms*R+atom1]=jnf(n,X1);
			if (n<-1) d_bessel[natoms*R+atom1]=powf(-1.0,(Real)n)*jnf(-n,X1);
		}//abs_n
	}//atom1 R	
}


__global__
void calculate_bessels_derivatives_kernel(
	int l, 
	int n, 
	int abs_n, 
	int const natoms,
	int const start_R_index, 
	int const legal_R_values, 
	float * d_layer_lines_R, Real * d_r, 
	float * d_bessel, 
	float * d_bessel_plus_1)
{
	unsigned int atom1 = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int R = blockDim.y * blockIdx.y + threadIdx.y;
	if (atom1 < natoms && R < legal_R_values) {
		Real Rinv(d_layer_lines_R[start_R_index+R]);
		Real x_factor( 2*M_PI*Rinv );
		Real X1 (x_factor*d_r[atom1]);
		if ( abs_n <= X1 +2 ) {
			if (n==0) d_bessel[natoms*R+atom1]=j0f(X1);
			if (n==1) d_bessel[natoms*R+atom1]=j1f(X1);
			if (n==-1) d_bessel[natoms*R+atom1]=-j1f(X1);
			if (n>1) d_bessel[natoms*R+atom1]=jnf(n,X1);
			if (n<-1) d_bessel[natoms*R+atom1]=powf(-1.0,(float)n)*jnf(-n,X1);
			//Calculating n+1 bessels
			int l = n+1;
			if (l==0) d_bessel_plus_1[natoms*R+atom1]=j0f(X1);
			if (l==1) d_bessel_plus_1[natoms*R+atom1]=j1f(X1);
			if (l==-1) d_bessel_plus_1[natoms*R+atom1]=-j1f(X1);
			if (l>1) d_bessel_plus_1[natoms*R+atom1]=jnf(l,X1);
			if (l<-1) d_bessel_plus_1[natoms*R+atom1]=powf(-1.0,(float)l)*jnf(-l,X1);
		}//abs_n
	}//atom1 R      
}


__global__
void calculate_phase_kernel(
	int l, 
	int n, 
	int abs_n, 
	int const natoms,
  float const c_, 
	Real * d_phi, 
	Real * d_z, 
	Real * d_r, 
	float * d_phase)
{
	//Main thread indexes for R, atom1, atom2
	unsigned int atom1 = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int atom2 = blockDim.y * blockIdx.y + threadIdx.y;

	if (atom1 < natoms && atom2 <= atom1) {
		d_phase[natoms*atom2+atom1] = (cosf(n*(d_phi[atom2]-d_phi[atom1])+2*M_PI*l/c_*(d_z[atom1]-d_z[atom2])));
	}
}


__global__
void calculate_phase_derivatives(
	int l, 
	int n, 
	int abs_n, 
	int const natoms,
 	float const c_,
	Real * d_phi,
	Real * d_z, 	
	Real * d_r,
	float * d_phase, 
	float * d_phase_prime)
{
	//Main thread indexes for R, atom1, atom2
	unsigned int atom1 = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int atom2 = blockDim.y * blockIdx.y + threadIdx.y;
	if (atom1 < natoms && atom2 <= atom1) {
		float phase_arg = n*(d_phi[atom2]-d_phi[atom1] )+2*M_PI*l/c_*( d_z[atom1]-d_z[atom2]);
		d_phase[natoms*atom2+atom1] = cosf(phase_arg);
		d_phase[natoms*atom1+atom2] = cosf(phase_arg);
		d_phase_prime[natoms*atom2+atom1] = -sinf(phase_arg);
		d_phase_prime[natoms*atom1+atom2] = sinf(phase_arg);
	}
}


__global__
void calculate_intensity_kernel(
	int l, 
	int n, 
	int abs_n, 
	int const natoms,
	int const legal_R_values,
	float const c_, 
	Size * d_atom_type_number,  
	float * d_layer_lines_R, 
	float * d_form_factors,
	Real * d_phi, 
	Real * d_z,
	Real * d_r, 
	float * d_bessel, 
	float * d_phase, 
	float * d_I) 
{
	//Main thread indexes for R, atom1, atom2
	unsigned int atom1 = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int atom2 = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int R = blockDim.z * blockIdx.z + threadIdx.z;
	
	if (atom1 < natoms && atom2 <= atom1 && R < legal_R_values) {
		//float d_phase(0);
		float Rinv(d_layer_lines_R[R]);
		float x_factor( 2*M_PI*Rinv );
		float X1 (x_factor*d_r[atom1]);
		float X2 (x_factor*d_r[atom2]);
		if ( abs_n <= X1 +2 && abs_n <= X2 +2 ) {
			int atom_type_index1 ( d_atom_type_number[atom1] );
			int atom_type_index2 ( d_atom_type_number[atom2] );
	       
			float f_atom1( d_form_factors[legal_R_values*(atom_type_index1-1)+R] );
			float f_atom2( d_form_factors[legal_R_values*(atom_type_index2-1)+R] );
			//__syncthreads();
			float dummyI( f_atom1*f_atom2*d_bessel[natoms*R+atom1]*d_bessel[natoms*R+atom2]*d_phase[natoms*atom2+atom1] );
			d_I[natoms*(natoms*R+atom1)+atom2]+= (atom1==atom2) ? dummyI : 2*dummyI;
		} //end abs_n condition
	} //end main if statement
}

//This function works as well but it is bit slower
__global__
void calculate_intensity_3d_kernel(int l, int n, int abs_n, int const natoms,
	int const legal_R_values, 
	float const c_, Size * d_atom_type_number,  
	float * d_layer_lines_R, float * d_form_factors,
	Real * d_phi, Real * d_z,
	Real * d_r, float * d_bessel, float * d_phase, float * d_I) 
{
	for (int atom1=blockIdx.x; atom1 < natoms; atom1 += gridDim.x){
		for (int R=threadIdx.y; R<legal_R_values; R+= blockDim.y){
			for (int atom2=threadIdx.x; atom2<=atom1; atom2 += blockDim.x){
      	float Rinv(d_layer_lines_R[R]);
      	float x_factor( 2*M_PI*Rinv );
      	float X1 (x_factor*d_r[atom1]);
      	float X2 (x_factor*d_r[atom2]);
      	if ( abs_n <= X1 +2 && abs_n <= X2 +2 ) {
					int atom_type_index1 ( d_atom_type_number[atom1] );
					int atom_type_index2 ( d_atom_type_number[atom2] );

					float f_atom1( d_form_factors[legal_R_values*(atom_type_index1-1)+R] );
					float f_atom2( d_form_factors[legal_R_values*(atom_type_index2-1)+R] );
					//__syncthreads();
					float dummyI( f_atom1*f_atom2*d_bessel[natoms*R+atom1]*d_bessel[natoms*R+atom2]*d_phase[natoms*atom2+atom1] );
					d_I[natoms*(natoms*R+atom1)+atom2]+= (atom1==atom2) ? dummyI : 2*dummyI;
        }
			}
		}
	}
}
//template< class T >
__global__
void calculate_derivatives_kernel(int l, int n, int abs_n, 
	int const natoms, int const start_R_index, 
	int const legal_R_values, int const max_R_values,
	float const scale_factor_, float const square_obs_,
	float const c_, Real * d_phi, Real * d_z, Real * d_r,
	float * d_layer_lines_R, float * d_layer_lines_I,
	float * d_form_factors, Size * d_atom_type_number,
	float * d_bessel, float * d_bessel_plus_1, float * d_I,
	float * d_phases, float * d_phases_prime, 
	float3 * D, float3 * D_cross_R,
	bool rfactor_refinement) 
{
        //Main thread indexes for R, atom1, atom2
	unsigned int atom1 = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int atom2 = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int R = blockDim.z * blockIdx.z + threadIdx.z;

	if (atom1 < natoms && atom2 < natoms && R < legal_R_values) {
		float Rinv(d_layer_lines_R[start_R_index+R]);
		float x_factor( 2*M_PI*Rinv );
		float X1 (x_factor*d_r[atom1]);
		float X2 (x_factor*d_r[atom2]);
		if ( abs_n <= X1 +2 && abs_n <= X2 +2 ) {	
			//Bessel values retreived
			float jn1( d_bessel[natoms*R+atom1] );
			float jn1_plus_1( d_bessel_plus_1[natoms*R+atom1] );
			float jn2( d_bessel[natoms*R+atom2] );
		
			float d_phase( d_phases[natoms*atom2+atom1] );
			float d_phase_prime( d_phases_prime[natoms*atom2+atom1] );
                	//__syncthreads();
			int atom_type_index1 ( d_atom_type_number[atom1] );
			float Jn_deriv_atom1( -x_factor*jn1_plus_1+n*jn1/d_r[atom1] );
			if ( fabs(d_r[atom1] ) < 1e-2 ) Jn_deriv_atom1=0.0f;
                                        
			float3 cartesian_coord_atom1 = {d_r[atom1]*cosf(d_phi[atom1]), d_r[atom1]*sinf(d_phi[atom1]), d_z[atom1] };
			float3 unit_r = { cosf(d_phi[atom1]), sinf(d_phi[atom1]), 0.0f };
			float3 unit_x = { 1.0f, 0.0f, 0.0f };
			float3 unit_z = { 0.0f, 0.0f, 1.0f };
			
			//Temporary kernel definition
			float3 D_ker= { 0.0f, 0.0f, 0.0f };
			float3 D_cross_R_ker = { 0.0f, 0.0f, 0.0f };
			
			if (atom1==atom2) {
				float tmp( 2*d_form_factors[max_R_values*(atom_type_index1-1)+start_R_index+R]\
						*d_form_factors[max_R_values*(atom_type_index1-1)+start_R_index+R]\
						*jn1*Jn_deriv_atom1 );
				D_ker = tmp*unit_r;
				D_cross_R_ker = cross(D_ker,cartesian_coord_atom1);
				//__syncthreads();
			}
			
			if (atom1!=atom2) {
				int atom_type_index2 ( d_atom_type_number[atom2] );
				float fact( d_form_factors[max_R_values*(atom_type_index1-1)+start_R_index+R]\
							*d_form_factors[max_R_values*(atom_type_index2-1)+start_R_index+R]*jn2 );

				float dr( Jn_deriv_atom1*fact*d_phase );
				float dphi( n*jn1*fact*d_phase_prime );                
				float dz ( 2*M_PI*l/c_*jn1*fact*d_phase_prime );
                		
				float3 D_tmp = { 0.0f,0.0f,0.0f };
				float3 dphi_vec = { 0.0f,0.0f,0.0f };
				if ( d_r[atom1] >= 1e-2 ) {
					if (fabs(sinf(d_phi[atom1])) > 1e-3)
 						dphi_vec = (unit_x-cosf(d_phi[atom1])*unit_r)/(sinf(d_phi[atom1])*d_r[atom1]);
					else 	{
						float3 dummy_dphi = {0.0f,(-1.0f/d_r[atom1]),0.0f};	
						dphi_vec = dummy_dphi;
					}
				}
				D_tmp = 2.0f*(dr*unit_r + dz*unit_z + dphi*dphi_vec);
				D_ker = D_ker+D_tmp;
				//TODO: Check if it is ok with math formula
				D_cross_R_ker=D_cross_R_ker+cross(D_tmp,cartesian_coord_atom1);
				//__syncthreads();
			}

			float dummy_factor(0); 
			if (rfactor_refinement) {
				if (d_I[start_R_index+R] > 0) {
					float F_diff (scale_factor_*sqrt(d_I[start_R_index+R]) - fabs(d_layer_lines_I[start_R_index+R]));
					dummy_factor = scale_factor_*F_diff/(2*square_obs_*fabs(F_diff)*sqrt(d_I[start_R_index+R])); 
				}
			} 
			else { 
				float dummy3( d_layer_lines_I[start_R_index+R]*d_layer_lines_I[start_R_index+R] );
				float I_diff( scale_factor_*d_I[start_R_index+R] - dummy3 );
				dummy_factor = 2*scale_factor_*I_diff/square_obs_;
			}
			D[natoms*(natoms*R+atom1)+atom2] += dummy_factor*D_ker;
			D_cross_R[natoms*(natoms*R+atom1)+atom2] += dummy_factor*D_cross_R_ker;
		} //end abs_n condition
	} //end main if statement
}


__global__
void sum_intensity_kernel(int const natoms, int const legal_R_values,
	float * d_I, float * d_I_R)
{
	//Main thread indexes for R, atom1, atom2
	unsigned int R = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int atom1 = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int atom2 = blockDim.z * blockIdx.z + threadIdx.z;
	if (atom1 < natoms && atom2 < natoms && atom2 <= atom1 && R < legal_R_values)
		atomicAdd(&d_I_R[R],d_I[natoms*(natoms*R+atom1)+atom2]);
}

__global__
void sum_intensity_3d_kernel(int const natoms, int const legal_R_values,
														float * d_I_l, float * d_I_R)
{  
	for (int R=blockIdx.x; R< legal_R_values; R += gridDim.x){
		float tmp_I_R(0);
		for (int atom1=threadIdx.y; atom1<natoms; atom1+= blockDim.y){
			for (int atom2=threadIdx.x; atom2<=atom1; atom2 += blockDim.x){
				tmp_I_R += d_I_l[natoms*(natoms*R+atom1)+atom2];
			}
		}
		atomicAdd(&d_I_R[R], tmp_I_R);
	}
}


__global__
void sum_derivatives_chi_3d_kernel(int const natoms,
	int const legal_R_values,
	float3 * d_dchi2_d, float3 * d_dchi2_d_cross_R, 
	float3 * D, float3 * D_cross_R)
{
    
    
	for (int atom1=blockIdx.x; atom1< natoms; atom1 += gridDim.x){
    float3 tmp_dchi2_d = { 0.0f, 0.0f, 0.0f };
    float3 tmp_dchi2_d_cross_R = { 0.0f, 0.0f, 0.0f };
    for (int R=threadIdx.y; R<legal_R_values; R+= blockDim.y){
			for (int atom2=threadIdx.x; atom2<natoms; atom2 += blockDim.x){
				tmp_dchi2_d += D[natoms*(natoms*R+atom1)+atom2];
				tmp_dchi2_d_cross_R+=D_cross_R[natoms*(natoms*R+atom1)+atom2];
			}
		}
		atomicAdd( &d_dchi2_d[atom1].x,tmp_dchi2_d.x );
		atomicAdd( &d_dchi2_d[atom1].y,tmp_dchi2_d.y );
		atomicAdd( &d_dchi2_d[atom1].z,tmp_dchi2_d.z );
		atomicAdd( &d_dchi2_d_cross_R[atom1].x,tmp_dchi2_d_cross_R.x );
		atomicAdd( &d_dchi2_d_cross_R[atom1].y,tmp_dchi2_d_cross_R.y );
		atomicAdd( &d_dchi2_d_cross_R[atom1].z,tmp_dchi2_d_cross_R.z );
	}
}

__global__
void sum_derivatives_chi_kernel(int const natoms, int const legal_R_values,
	float3 * d_dchi2_d, float3 * d_dchi2_d_cross_R, 
	float3 * D, float3 * D_cross_R)
{
	//Main thread indexes for R, atom1, atom2
	unsigned int atom1 = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int atom2 = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int R = blockDim.z * blockIdx.z + threadIdx.z;

	if (atom1 < natoms && atom2 < natoms && R < legal_R_values) {
		atomicAdd( &d_dchi2_d[atom1].x,D[natoms*(natoms*R+atom1)+atom2].x );
		atomicAdd( &d_dchi2_d[atom1].y,D[natoms*(natoms*R+atom1)+atom2].y );
		atomicAdd( &d_dchi2_d[atom1].z,D[natoms*(natoms*R+atom1)+atom2].z );
		atomicAdd( &d_dchi2_d_cross_R[atom1].x,D_cross_R[natoms*(natoms*R+atom1)+atom2].x );
		atomicAdd( &d_dchi2_d_cross_R[atom1].y,D_cross_R[natoms*(natoms*R+atom1)+atom2].y );
		atomicAdd( &d_dchi2_d_cross_R[atom1].z,D_cross_R[natoms*(natoms*R+atom1)+atom2].z );
	}
}


// Utility function for checking CUDA runtime API results
// can be wrapped around any runtime API call.
void checkCuda(hipError_t result)
{
	if (result != hipSuccess) {
		TR.Error<<"CUDA Runtime Error: "<<hipGetErrorString(result);
		utility::exit( EXIT_FAILURE, __FILE__, __LINE__);
  }
}

void  calculate_intensity_gpu(
	Size const l_max,
	Size const natoms,
	utility::vector0 < utility::vector0 < int > >::iterator & nvals,
	utility::vector0< utility::vector1< Real > >::iterator & layer_lines_R,	
	utility::vector0< utility::vector1< Real > > & I,
	utility::vector0< utility::vector1< utility::vector1< Real > > >::iterator & form_factors,
	utility::vector1< Real > & phi,
	utility::vector1< Real > & z,
	utility::vector1< Real > & r,
	utility::vector1< Size > & atom_type_number,
	Real const c_, 
	Real const res_cutoff_low_, 
	Real const res_cutoff_high_,
	int const gpu_processor_)
{
	//Cuda error retruned by checkCuda function
	hipError_t error;
	
	checkCuda ( hipSetDevice( gpu_processor_ ) );
	//TODO: observe!
	//checkCuda ( hipDeviceReset() );
	
	//GPU device variables 
	Real * d_phi;
	Real * d_z;
	Real * d_r;
	Size * d_atom_type_number;


	//Input variables initialized by H2D memcpy. 
	checkCuda ( hipMalloc((void **)&d_phi, sizeof( Real ) * natoms) );
	checkCuda ( hipMalloc((void **)&d_z, sizeof( Real ) * natoms) );
	checkCuda ( hipMalloc((void **)&d_atom_type_number, sizeof( Size ) *  natoms) );
	checkCuda ( hipMalloc((void **)&d_r, sizeof( Real ) * natoms) );

	//Copying values from CPU vectors to previously initialized variables on the GPU
	checkCuda ( hipMemcpy(d_phi, &phi[ 1 ], sizeof( Real ) * natoms, hipMemcpyHostToDevice) );
	checkCuda ( hipMemcpy(d_z, &z[ 1 ], sizeof( Real ) * natoms, hipMemcpyHostToDevice) );
	checkCuda ( hipMemcpy(d_r, &r[ 1 ], sizeof( Real ) * natoms, hipMemcpyHostToDevice) );
	checkCuda ( hipMemcpy(d_atom_type_number, &atom_type_number[ 1 ], sizeof( Size ) * natoms, hipMemcpyHostToDevice) );

	float totalElapsedTime_bessel(0);
	float totalElapsedTime_main(0);
	float totalElapsedTime_reduct(0);
	float totalElapsedTime_phase(0);
	float Cycle;
	hipEvent_t t1_bessel,t2_bessel, t1_main, t2_main, t1_reduct, t2_reduct, t1_phase, t2_phase;
	hipEventCreate(&t1_bessel);
	hipEventCreate(&t2_bessel);
	hipEventCreate(&t1_main);
	hipEventCreate(&t2_main);
	hipEventCreate(&t1_reduct);
	hipEventCreate(&t2_reduct);
	hipEventCreate(&t1_phase);
	hipEventCreate(&t2_phase);
        
        
	for ( Size l=0; l <= l_max; ++l ) {
		
		Size max_b_order( nvals[l].size() );
		Size max_R_values( layer_lines_R[l].size());
                
		//Intensity per each layer line
		float * h_I_R;
		float * h_layer_lines_R_l;
		float * h_form_factors_l;
             
		if (max_R_values==0) continue;
                
		h_I_R =  (float * ) malloc( max_R_values * sizeof( float ));
		h_layer_lines_R_l =  ( float * ) malloc(max_R_values *  sizeof( float ));
		h_form_factors_l =  ( float * ) malloc(5 * max_R_values *  sizeof( float ));	
		
		Size t_count(0);		
		for ( Size atom=0; atom<5; ++atom ) {
			t_count = 0;
			for ( Size R=0; R<max_R_values; ++R ) {
				h_form_factors_l[max_R_values*atom+t_count]=form_factors[l][atom+1][R+1];
				t_count++;
			}
		}
		t_count =0;	
		for ( Size R=0; R< max_R_values; ++R ) {
			h_layer_lines_R_l[t_count]=layer_lines_R[l][R+1];
			t_count++;
		}
		
		//GPU device variables. It's bit faster when using float instead of Real	
		float * d_I_l;
		float * d_I_R;
		float * d_layer_lines_R_l;
		float * d_form_factors_l;
		float * d_bessels;
		float * d_phases;
	
		checkCuda ( hipMalloc((void **)&d_layer_lines_R_l, sizeof( float ) * max_R_values) );
		checkCuda ( hipMalloc((void **)&d_form_factors_l, sizeof( float ) * 5 * max_R_values) );
		checkCuda ( hipMemcpy(d_layer_lines_R_l, h_layer_lines_R_l, sizeof( float ) * max_R_values, hipMemcpyHostToDevice) );
		checkCuda ( hipMemcpy(d_form_factors_l, h_form_factors_l, sizeof( float ) * 5 * max_R_values, hipMemcpyHostToDevice) );	
		
		checkCuda ( hipMalloc((void **)&d_bessels, sizeof( float ) * natoms * max_R_values) );
		checkCuda ( hipMalloc((void **)&d_phases, sizeof( float ) * natoms * natoms) );
		
		//Output intensity
		checkCuda ( hipMalloc((void **)&d_I_l, sizeof( float ) * max_R_values * natoms * natoms ) );
		checkCuda ( hipMemset(d_I_l, 0.0, max_R_values * natoms * natoms * sizeof( float )));    
		checkCuda ( hipMalloc((void **)&d_I_R, sizeof( float ) * max_R_values) );
		checkCuda ( hipMemset(d_I_R, 0.0, max_R_values * sizeof( float )));
		//hipDeviceSynchronize();
		
               
		for ( Size b_order=1; b_order <= max_b_order; ++b_order ) {

			int n( nvals[l][b_order-1] );
			int abs_n( abs(n) );

			hipEventRecord(t1_phase,0);
			//Bessel function calculation (these values are precalcualated and store in GPU mem)
			dim3 threadsPerBlockPhase(8,16);
			dim3 numBlocksPhase((natoms+threadsPerBlockPhase.x-1)/threadsPerBlockPhase.x,
                     (natoms+threadsPerBlockPhase.y-1)/threadsPerBlockPhase.y);
			calculate_phase_kernel<<<numBlocksPhase, threadsPerBlockPhase >>>(l, n, abs_n, natoms, c_,
													d_phi, d_z, d_r, d_phases);
     error = hipGetLastError();
     if ( error != 0 ) {
				TR.Error<<"Problem running phase kernel! "<<hipGetErrorString(error)<<std::endl;
				utility::exit( EXIT_FAILURE, __FILE__, __LINE__);
			}

			hipEventRecord(t2_phase,0);
			hipEventSynchronize(t2_phase);
			hipEventElapsedTime(&Cycle,t1_phase,t2_phase);
			totalElapsedTime_phase +=Cycle;

			hipEventRecord(t1_bessel,0);

			//Bessel function calculation (these values are precalcualated and store in GPU mem)
			dim3 threadsPerBlockBessel(8,16);
			dim3 numBlocksBessel((natoms+threadsPerBlockBessel.x-1)/threadsPerBlockBessel.x,
			(max_R_values+threadsPerBlockBessel.y-1)/threadsPerBlockBessel.y);
			calculate_bessels_kernel<<<numBlocksBessel, threadsPerBlockBessel>>>(l, n, abs_n, natoms,
                                                                       	max_R_values, d_layer_lines_R_l,
                                                                        d_r, d_bessels);
			
			error = hipGetLastError();
			if ( error != 0 ) {
				TR.Error<<"Problem running bessel kernel! "<<hipGetErrorString(error)<<std::endl; 
				utility::exit( EXIT_FAILURE, __FILE__, __LINE__);
			}
			//////////////////
			hipEventRecord(t2_bessel,0);
			hipEventSynchronize(t2_bessel);
			hipEventElapsedTime(&Cycle,t1_bessel,t2_bessel);
			totalElapsedTime_bessel +=Cycle;
			/////////////////
			hipEventRecord(t1_main,0);

			//Main GPU function - calculating intensity			
			dim3 threadsPerBlock(2, 8, 8);
			dim3 numBlocks((natoms+threadsPerBlock.x-1)/threadsPerBlock.x,
				(natoms+threadsPerBlock.y-1)/threadsPerBlock.y,
				(max_R_values+threadsPerBlock.z-1)/threadsPerBlock.z);

			calculate_intensity_kernel<<<numBlocks, threadsPerBlock>>>(l, n, abs_n, natoms,
									max_R_values,
									c_, d_atom_type_number, d_layer_lines_R_l, 
									 d_form_factors_l,
									d_phi, d_z, d_r, d_bessels, d_phases, d_I_l);
                        
			error = hipGetLastError();
			if ( error != 0 ) {
				TR.Error<<"Problem running intensity kernel! "<<hipGetErrorString(error)<<std::endl; 
				utility::exit( EXIT_FAILURE, __FILE__, __LINE__);
			}

			hipEventRecord(t2_main,0);
			hipEventSynchronize(t2_main);
			hipEventElapsedTime(&Cycle,t1_main,t2_main);
			totalElapsedTime_main +=Cycle;
		}//end b_order
                
		hipEventRecord(t1_reduct,0);
                
		//Horizontal and vertical summing of intensity
		/*dim3 threadsPerBlockSum(32, 4, 4);
		dim3 numBlocksSum((legal_R_values+threadsPerBlockSum.x-1)/threadsPerBlockSum.x,
		(natoms+threadsPerBlockSum.y-1)/threadsPerBlockSum.y,
		(natoms+threadsPerBlockSum.z-1)/threadsPerBlockSum.z);*/

		dim3 threadsPerBlockSum(8, 64);
		dim3 numBlocksSum(max_R_values);
		sum_intensity_3d_kernel<<<numBlocksSum, threadsPerBlockSum>>>(natoms, max_R_values, d_I_l, d_I_R);
		error = hipGetLastError();
		if ( error != 0 ) {
			TR.Error<<"Problem running kernel! "<<hipGetErrorString(error)<<std::endl; 
			utility::exit( EXIT_FAILURE, __FILE__, __LINE__);
		}
    
    hipEventRecord(t2_reduct,0);
    hipEventSynchronize(t2_reduct);
    hipEventElapsedTime(&Cycle,t1_reduct,t2_reduct);
    totalElapsedTime_reduct +=Cycle;
                
		checkCuda( hipMemcpy(h_I_R, d_I_R ,max_R_values*sizeof(float),hipMemcpyDeviceToHost) );
		hipDeviceSynchronize();
		t_count =0;
		for ( Size R=0; R< max_R_values; ++R ) {
			I[l][R+1] = h_I_R[t_count];
			t_count++;
		}		

		hipFree( d_layer_lines_R_l );
		hipFree( d_form_factors_l );
		hipFree( d_I_l );
		hipFree( d_I_R );
		hipFree( d_bessels );
		hipFree( d_phases );
		free(h_I_R);
		free(h_layer_lines_R_l);
		free(h_form_factors_l);
	} //end l
	
	TR << " Phase, bessel,  main, reduction times " <<totalElapsedTime_phase \
                <<", "<<totalElapsedTime_bessel<<", "<< totalElapsedTime_main \
                <<", "<< totalElapsedTime_reduct<< " ms.\n";
	//Freeing memory for the device variables
	hipFree( d_phi );
	hipFree( d_z );
	hipFree( d_r);
	hipFree( d_atom_type_number );
	//free(streams);
}

void  calculate_derivatives_gpu(
	Size const l_max,
	Size const natoms,
	utility::vector0< utility::vector0 < int > >::iterator & nvals,
	utility::vector0< utility::vector1< Real > >::iterator & layer_lines_R,
	utility::vector0< utility::vector1< Real > >::iterator & layer_lines_I,
	utility::vector0< utility::vector1< Real > > & I,
	utility::vector0< utility::vector1< utility::vector1< Real > > >::iterator & form_factors,
	utility::vector1< Real > & phi,
	utility::vector1< Real > & z,
	utility::vector1< Real > & r,
	utility::vector1< Size > & atom_type_number,
	utility::vector1< numeric::xyzVector< core::Real > > & dchi2_d, 
	utility::vector1< numeric::xyzVector< core::Real > > & dchi2_d_cross_R,
	Real const c_,
	Real const res_cutoff_low_,
	Real const res_cutoff_high_,
	Real const scale_factor_, 
	Real const square_obs_,
	int const gpu_processor_,
	bool rfactor_refinement)
{
	//Cuda error retruned by checkCuda function
	hipError_t error;
	checkCuda ( hipSetDevice( gpu_processor_ ) );        

	//It is not necessary to reset but if there is sth wrong the it will raise up an error
	checkCuda ( hipDeviceReset() );

	//GPU device variables 
	Real * d_phi;
	Real * d_z;
	Real * d_r;
	Size * d_atom_type_number;

	//Input variables initialized by H2D memcpy. 
	checkCuda ( hipMalloc((void **)&d_phi, sizeof( Real ) * natoms) );
	checkCuda ( hipMalloc((void **)&d_z, sizeof( Real ) * natoms) );
	checkCuda ( hipMalloc((void **)&d_atom_type_number, sizeof( Size ) *  natoms) );
	checkCuda ( hipMalloc((void **)&d_r, sizeof( Real ) * natoms) );

	//Copying values from CPU vectors to previously initialized variables on the GPU
	checkCuda ( hipMemcpy(d_phi, &phi[ 1 ], sizeof( Real ) * natoms, hipMemcpyHostToDevice) );
	checkCuda ( hipMemcpy(d_z, &z[ 1 ], sizeof( Real ) * natoms, hipMemcpyHostToDevice) );
	checkCuda ( hipMemcpy(d_r, &r[ 1 ], sizeof( Real ) * natoms, hipMemcpyHostToDevice) );
	checkCuda ( hipMemcpy(d_atom_type_number, &atom_type_number[ 1 ], sizeof( Size ) * natoms, hipMemcpyHostToDevice) );

	float totalElapsedTime_reduct(0);
	float Cycle;
	hipEvent_t t1_reduct, t2_reduct;
	hipEventCreate(&t1_reduct);
	hipEventCreate(&t2_reduct);

	for ( Size l=0; l <= l_max; ++l ) {
		Size max_b_order( nvals[l].size() );
		Size max_R_values( layer_lines_R[l].size());
		
		//d_dchi and d_dchi_cross_R for each l
		float3 * h_dchi_l;
		float3 * h_dchi_cross_R_l;
		h_dchi_l = (float3 * ) malloc( natoms * sizeof( float3 ));
		h_dchi_cross_R_l = (float3 * ) malloc( natoms * sizeof( float3 ));

		//Intensity per each layer line
		float * h_I_l;
		float * h_layer_lines_R_l;
		float * h_layer_lines_I_l;
		float * h_form_factors_l;
                
		if (max_R_values==0) continue;
                
		h_I_l =  (float * ) malloc( max_R_values * sizeof( float ));
		h_layer_lines_R_l =  ( float * ) malloc(max_R_values *  sizeof( float ));
		h_layer_lines_I_l =  ( float * ) malloc(max_R_values *  sizeof( float ));
		h_form_factors_l =  ( float * ) malloc(5 * max_R_values *  sizeof( float ));

		Size t_count(0);
		for ( Size atom=0; atom<5; ++atom ) {
        t_count = 0;
        for ( Size R=0; R<max_R_values; ++R ) {
					h_form_factors_l[max_R_values*atom+t_count]=form_factors[l][atom+1][R+1];
					t_count++;
        }
		}
		t_count =0;
		for ( Size R=0; R< max_R_values; ++R ) {
			h_layer_lines_R_l[t_count]=layer_lines_R[l][R+1];
			h_layer_lines_I_l[t_count]=layer_lines_I[l][R+1];
			h_I_l[t_count] = I[l][R+1];
			t_count++;
		}
		
		//GPU device variables. It's bit faster when using float instead of Real        
		float3 * d_dchi_l;
		float3 * d_dchi_cross_R_l;
		float * d_I_l;
		float * d_layer_lines_R_l;
		float * d_layer_lines_I_l;
 		float * d_form_factors_l;
		float * d_phases;
		float * d_phases_prime;

		checkCuda ( hipMalloc((void **)&d_layer_lines_R_l, sizeof( float ) * max_R_values) );
		checkCuda ( hipMalloc((void **)&d_layer_lines_I_l, sizeof( float ) * max_R_values) );
		checkCuda ( hipMalloc((void **)&d_I_l, sizeof( float ) * max_R_values) );
		checkCuda ( hipMalloc((void **)&d_form_factors_l, sizeof( float ) * 5 * max_R_values) );
		checkCuda ( hipMemcpy(d_layer_lines_R_l, h_layer_lines_R_l, sizeof( float ) * max_R_values, hipMemcpyHostToDevice) );
		checkCuda ( hipMemcpy(d_layer_lines_I_l, h_layer_lines_I_l, sizeof( float ) * max_R_values, hipMemcpyHostToDevice) );
		checkCuda ( hipMemcpy(d_I_l, h_I_l, sizeof( float ) * max_R_values, hipMemcpyHostToDevice) );
		checkCuda ( hipMemcpy(d_form_factors_l, h_form_factors_l, sizeof( float ) * 5 * max_R_values, hipMemcpyHostToDevice) );

		checkCuda ( hipMalloc((void **)&d_phases, sizeof( float ) * natoms * natoms) );
		checkCuda ( hipMalloc((void **)&d_phases_prime, sizeof( float ) * natoms * natoms) );

		checkCuda ( hipMalloc((void **)&d_dchi_l, sizeof( float3 ) * natoms ) );
		checkCuda ( hipMemset(d_dchi_l, 0.0, natoms * sizeof( float3 )));
		checkCuda ( hipMalloc((void **)&d_dchi_cross_R_l, sizeof( float3 ) * natoms) );
		checkCuda ( hipMemset(d_dchi_cross_R_l, 0.0, natoms * sizeof( float3 )));

		hipDeviceSynchronize();
	
		size_t gpu_mem_tot = 0;
		size_t gpu_mem_free = 0;
		hipMemGetInfo(&gpu_mem_free, &gpu_mem_tot) ;
		error = hipGetLastError();
    if ( error != 0 ) {
				TR.Error<<"Cannot check memory! "<<hipGetErrorString(error)<<std::endl; 
				utility::exit( EXIT_FAILURE, __FILE__, __LINE__);
		}
		//Chunk size is set based on R values, 500MB and memory for bessels are left as a buffer and 2 variables has to be allocated 
		Size chunk_size =float(gpu_mem_free - 2*sizeof( float ) * natoms * max_R_values - 524288000)/float((sizeof( float3 ) * natoms * natoms ))/2 ;
		Size number_of_chunks(ceil(float(max_R_values)/float(chunk_size)));
		Size start_R_index;
		Size legal_R_values(floor(float(max_R_values)/float(number_of_chunks)));
		for (Size chunk=0; chunk<number_of_chunks; ++chunk) {
			start_R_index = chunk*legal_R_values;
			//Last chunk takes what is left
			if (chunk==number_of_chunks-1) legal_R_values = max_R_values - chunk*legal_R_values; 
			//D and D_cross_R init are splitted into chunks depending on memory limits
			float * d_bessels;
			float * d_bessels_plus_1;	
			float3 * d_D_l;
			float3 * d_D_cross_R_l;
			
			checkCuda ( hipMalloc((void **)&d_bessels, sizeof( float ) * natoms * legal_R_values) );
			checkCuda ( hipMalloc((void **)&d_bessels_plus_1, sizeof( float ) * natoms * legal_R_values) );
			checkCuda ( hipMalloc((void **)&d_D_l, sizeof( float3 ) * legal_R_values * natoms * natoms ) );
			checkCuda ( hipMemset(d_D_l, 0.0, legal_R_values * natoms * natoms * sizeof( float3 )));
			checkCuda ( hipMalloc((void **)&d_D_cross_R_l, sizeof( float3 ) * natoms * natoms * legal_R_values) );
			checkCuda ( hipMemset(d_D_cross_R_l, 0.0, legal_R_values * natoms * natoms * sizeof( float3 )));

			for ( Size b_order=1; b_order <= max_b_order; ++b_order ) {
				int n( nvals[l][b_order-1] );
				int abs_n( abs(n) );

				//TODO: Phase and bessels might be done outside the chunk loop. Time benfit might be slight but memory required
				dim3 threadsPerBlockPhase(8,16);
				dim3 numBlocksPhase((natoms+threadsPerBlockPhase.x-1)/threadsPerBlockPhase.x,
        (natoms+threadsPerBlockPhase.y-1)/threadsPerBlockPhase.y);
				calculate_phase_derivatives<<<numBlocksPhase, threadsPerBlockPhase>>>(l, n, abs_n, natoms, c_,
                                                        d_phi, d_z, d_r,
                                                        d_phases, d_phases_prime);
				hipDeviceSynchronize();
				error = hipGetLastError();
				if ( error != 0 ) {
					TR.Error<<"Problem running kernel! "<<hipGetErrorString(error)<<std::endl; 
					utility::exit( EXIT_FAILURE, __FILE__, __LINE__);
				}

				//Bessel function calculation (these values are precalcualated and store in GPU mem)
				dim3 threadsPerBlockBessel(8,16);
				dim3 numBlocksBessel((natoms+threadsPerBlockBessel.x-1)/threadsPerBlockBessel.x,
        (legal_R_values+threadsPerBlockBessel.y-1)/threadsPerBlockBessel.y);
				calculate_bessels_derivatives_kernel<<<numBlocksBessel, threadsPerBlockBessel>>>(l, n, abs_n, natoms,
                                        start_R_index, legal_R_values, d_layer_lines_R_l,
                                        d_r, d_bessels, d_bessels_plus_1);
				hipDeviceSynchronize();
				error = hipGetLastError();
				if ( error != 0 ) {
					TR.Error<<"Problem running kernel! "<<hipGetErrorString(error)<<std::endl; 
					utility::exit( EXIT_FAILURE, __FILE__, __LINE__);
				}

				//Main GPU function - calculating intensity                     
				dim3 threadsPerBlock(2, 8, 8);
				dim3 numBlocks((natoms+threadsPerBlock.x-1)/threadsPerBlock.x,
      			(natoms+threadsPerBlock.y-1)/threadsPerBlock.y,
      			(legal_R_values+threadsPerBlock.z-1)/threadsPerBlock.z);

				calculate_derivatives_kernel<<<numBlocks, threadsPerBlock>>>(l, n, abs_n, natoms, 
									start_R_index, legal_R_values, max_R_values,
									scale_factor_, square_obs_,
									c_, d_phi, d_z, d_r,
									d_layer_lines_R_l, d_layer_lines_I_l,
									d_form_factors_l, d_atom_type_number,
									d_bessels, d_bessels_plus_1, d_I_l,
									d_phases, d_phases_prime,
									d_D_l, d_D_cross_R_l,
									rfactor_refinement);
									//d_dchi_all, d_dchi_cross_R_all);
				hipDeviceSynchronize();
				error = hipGetLastError();
				if ( error != 0 ) {
					TR.Error<<"Problem running main kernel! "<<hipGetErrorString(error)<<std::endl; 
					utility::exit( EXIT_FAILURE, __FILE__, __LINE__);
				}

			}//end b_order
			//Horizontal and vertical summing of intensity
			hipEventRecord(t1_reduct,0);

			dim3 threadsPerBlockSum(16, 8);
			dim3 numBlocksSum((natoms+threadsPerBlockSum.x-1)/threadsPerBlockSum.x);
			//dim3 numBlocksSum(natoms);
			sum_derivatives_chi_3d_kernel<<<numBlocksSum,threadsPerBlockSum>>>(natoms, legal_R_values, 
					d_dchi_l, d_dchi_cross_R_l, d_D_l, d_D_cross_R_l);
			hipDeviceSynchronize();
			error = hipGetLastError();
			if ( error != 0 ) {
				TR.Error<<"Problem running summation kernel! "<<hipGetErrorString(error)<<std::endl; 
				utility::exit( EXIT_FAILURE, __FILE__, __LINE__);
			}

			hipEventRecord(t2_reduct,0);
			hipEventSynchronize(t2_reduct);
			hipEventElapsedTime(&Cycle,t1_reduct,t2_reduct);
			totalElapsedTime_reduct +=Cycle;
			hipFree( d_D_l );
			hipFree( d_D_cross_R_l );
			hipFree( d_bessels );
			hipFree( d_bessels_plus_1 );
		}//end chunks
		checkCuda( hipMemcpy(h_dchi_l, d_dchi_l ,natoms*sizeof(float3),hipMemcpyDeviceToHost) );
		checkCuda( hipMemcpy(h_dchi_cross_R_l, d_dchi_cross_R_l, natoms*sizeof(float3),hipMemcpyDeviceToHost) );
             
		for ( Size atom1=0; atom1< natoms; ++atom1 ) {
			numeric::xyzVector< core::Real> dummy_dchi( h_dchi_l[atom1].x, h_dchi_l[atom1].y, h_dchi_l[atom1].z);
			numeric::xyzVector< core::Real> dummy_dchi_cross_R( h_dchi_cross_R_l[atom1].x, h_dchi_cross_R_l[atom1].y, h_dchi_cross_R_l[atom1].z );
			dchi2_d[atom1+1] += dummy_dchi;
			dchi2_d_cross_R[atom1+1] += dummy_dchi_cross_R; 
		}
	
		hipFree( d_layer_lines_R_l );
		hipFree( d_layer_lines_I_l );
		hipFree( d_form_factors_l );
		hipFree( d_I_l );
		hipFree( d_dchi_l );
		hipFree( d_dchi_cross_R_l );
		hipFree( d_phases );
		hipFree( d_phases_prime );
		free(h_dchi_l);
		free(h_dchi_cross_R_l);
		free(h_I_l);
		free(h_layer_lines_R_l);
		free(h_form_factors_l);
	}//end l
	TR << "reduction times " << totalElapsedTime_reduct<< " ms.\n";
	hipFree( d_phi);
	hipFree( d_r);
	hipFree( d_z);
	hipFree( d_atom_type_number );
}

} // namespace fiber_diffraction
} // namespace scoring
} // namespace core

